#include "hip/hip_runtime.h"
#include<cstdio>
#include<stdio.h>
#include<time.h>
#include<string.h>
#include<unistd.h>
#include<stdlib.h>


unsigned char *gpu_input_data_s,*gpu_output_data_s;
unsigned int *gpu_offset;
#define FINGERPRINT_LEN 20
#define MAX_CHUNK_LEN (16384)

#define MJW

#ifdef MJW

typedef struct
{
    unsigned long total[2];     /*!< number of bytes processed  */
    unsigned long state[5];     /*!< intermediate digest state  */
    unsigned char buffer[64];   /*!< data block being processed */

    unsigned char ipad[64];     /*!< HMAC: inner padding        */
    unsigned char opad[64];     /*!< HMAC: outer padding        */
}
sha1_context;

#ifndef GET_ULONG_BE
#define GET_ULONG_BE(n,b,i)                             \
{                                                       \
    (n) = ( (unsigned long) (b)[(i)    ] << 24 )        \
        | ( (unsigned long) (b)[(i) + 1] << 16 )        \
        | ( (unsigned long) (b)[(i) + 2] <<  8 )        \
        | ( (unsigned long) (b)[(i) + 3]       );       \
}
#endif

#ifndef PUT_ULONG_BE
#define PUT_ULONG_BE(n,b,i)                             \
{                                                       \
    (b)[(i)    ] = (unsigned char) ( (n) >> 24 );       \
    (b)[(i) + 1] = (unsigned char) ( (n) >> 16 );       \
    (b)[(i) + 2] = (unsigned char) ( (n) >>  8 );       \
    (b)[(i) + 3] = (unsigned char) ( (n)       );       \
}
#endif
/*
 * SHA-1 context setup
 */
__device__ void sha1_starts( sha1_context *ctx )
{
    ctx->total[0] = 0;
    ctx->total[1] = 0;

    ctx->state[0] = 0x67452301;
    ctx->state[1] = 0xEFCDAB89;
    ctx->state[2] = 0x98BADCFE;
    ctx->state[3] = 0x10325476;
    ctx->state[4] = 0xC3D2E1F0;
}

__device__ static void sha1_process( sha1_context *ctx, unsigned char data[64] )
{
    unsigned long temp, W[16], A, B, C, D, E;

    GET_ULONG_BE( W[ 0], data,  0 );
    GET_ULONG_BE( W[ 1], data,  4 );
    GET_ULONG_BE( W[ 2], data,  8 );
    GET_ULONG_BE( W[ 3], data, 12 );
    GET_ULONG_BE( W[ 4], data, 16 );
    GET_ULONG_BE( W[ 5], data, 20 );
    GET_ULONG_BE( W[ 6], data, 24 );
    GET_ULONG_BE( W[ 7], data, 28 );
    GET_ULONG_BE( W[ 8], data, 32 );
    GET_ULONG_BE( W[ 9], data, 36 );
    GET_ULONG_BE( W[10], data, 40 );
    GET_ULONG_BE( W[11], data, 44 );
    GET_ULONG_BE( W[12], data, 48 );
    GET_ULONG_BE( W[13], data, 52 );
    GET_ULONG_BE( W[14], data, 56 );
    GET_ULONG_BE( W[15], data, 60 );

#define S(x,n) ((x << n) | ((x & 0xFFFFFFFF) >> (32 - n)))

#define R(t)                                            \
(                                                       \
    temp = W[(t -  3) & 0x0F] ^ W[(t - 8) & 0x0F] ^     \
           W[(t - 14) & 0x0F] ^ W[ t      & 0x0F],      \
    ( W[t & 0x0F] = S(temp,1) )                         \
)

#define P(a,b,c,d,e,x)                                  \
{                                                       \
    e += S(a,5) + F(b,c,d) + K + x; b = S(b,30);        \
}

    A = ctx->state[0];
    B = ctx->state[1];
    C = ctx->state[2];
    D = ctx->state[3];
    E = ctx->state[4];

#define F(x,y,z) (z ^ (x & (y ^ z)))
#define K 0x5A827999

    P( A, B, C, D, E, W[0]  );
    P( E, A, B, C, D, W[1]  );
    P( D, E, A, B, C, W[2]  );
    P( C, D, E, A, B, W[3]  );
    P( B, C, D, E, A, W[4]  );
    P( A, B, C, D, E, W[5]  );
    P( E, A, B, C, D, W[6]  );
    P( D, E, A, B, C, W[7]  );
    P( C, D, E, A, B, W[8]  );
    P( B, C, D, E, A, W[9]  );
    P( A, B, C, D, E, W[10] );
    P( E, A, B, C, D, W[11] );
    P( D, E, A, B, C, W[12] );
    P( C, D, E, A, B, W[13] );
    P( B, C, D, E, A, W[14] );
    P( A, B, C, D, E, W[15] );
    P( E, A, B, C, D, R(16) );
    P( D, E, A, B, C, R(17) );
    P( C, D, E, A, B, R(18) );
    P( B, C, D, E, A, R(19) );

#undef K
#undef F

#define F(x,y,z) (x ^ y ^ z)
#define K 0x6ED9EBA1

    P( A, B, C, D, E, R(20) );
    P( E, A, B, C, D, R(21) );
    P( D, E, A, B, C, R(22) );
    P( C, D, E, A, B, R(23) );
    P( B, C, D, E, A, R(24) );
    P( A, B, C, D, E, R(25) );
    P( E, A, B, C, D, R(26) );
    P( D, E, A, B, C, R(27) );
    P( C, D, E, A, B, R(28) );
    P( B, C, D, E, A, R(29) );
    P( A, B, C, D, E, R(30) );
    P( E, A, B, C, D, R(31) );
    P( D, E, A, B, C, R(32) );
    P( C, D, E, A, B, R(33) );
    P( B, C, D, E, A, R(34) );
    P( A, B, C, D, E, R(35) );
    P( E, A, B, C, D, R(36) );
    P( D, E, A, B, C, R(37) );
    P( C, D, E, A, B, R(38) );
    P( B, C, D, E, A, R(39) );

#undef K
#undef F

#define F(x,y,z) ((x & y) | (z & (x | y)))
#define K 0x8F1BBCDC

    P( A, B, C, D, E, R(40) );
    P( E, A, B, C, D, R(41) );
    P( D, E, A, B, C, R(42) );
    P( C, D, E, A, B, R(43) );
    P( B, C, D, E, A, R(44) );
    P( A, B, C, D, E, R(45) );
    P( E, A, B, C, D, R(46) );
    P( D, E, A, B, C, R(47) );
    P( C, D, E, A, B, R(48) );
    P( B, C, D, E, A, R(49) );
    P( A, B, C, D, E, R(50) );
    P( E, A, B, C, D, R(51) );
    P( D, E, A, B, C, R(52) );
    P( C, D, E, A, B, R(53) );
    P( B, C, D, E, A, R(54) );
    P( A, B, C, D, E, R(55) );
    P( E, A, B, C, D, R(56) );
    P( D, E, A, B, C, R(57) );
    P( C, D, E, A, B, R(58) );
    P( B, C, D, E, A, R(59) );

#undef K
#undef F

#define F(x,y,z) (x ^ y ^ z)
#define K 0xCA62C1D6

    P( A, B, C, D, E, R(60) );
    P( E, A, B, C, D, R(61) );
    P( D, E, A, B, C, R(62) );
    P( C, D, E, A, B, R(63) );
    P( B, C, D, E, A, R(64) );
    P( A, B, C, D, E, R(65) );
    P( E, A, B, C, D, R(66) );
    P( D, E, A, B, C, R(67) );
    P( C, D, E, A, B, R(68) );
    P( B, C, D, E, A, R(69) );
    P( A, B, C, D, E, R(70) );
    P( E, A, B, C, D, R(71) );
    P( D, E, A, B, C, R(72) );
    P( C, D, E, A, B, R(73) );
    P( B, C, D, E, A, R(74) );
    P( A, B, C, D, E, R(75) );
    P( E, A, B, C, D, R(76) );
    P( D, E, A, B, C, R(77) );
    P( C, D, E, A, B, R(78) );
    P( B, C, D, E, A, R(79) );

#undef K
#undef F

    ctx->state[0] += A;
    ctx->state[1] += B;
    ctx->state[2] += C;
    ctx->state[3] += D;
    ctx->state[4] += E;
}

/*
 * SHA-1 process buffer
 */
__device__ void sha1_update( sha1_context *ctx, unsigned char *input, int ilen )
{
    int fill;
    unsigned long left;

    if( ilen <= 0 )
        return;

    left = ctx->total[0] & 0x3F;
    fill = 64 - left;

    ctx->total[0] += ilen;
    ctx->total[0] &= 0xFFFFFFFF;

    if( ctx->total[0] < (unsigned long) ilen )
        ctx->total[1]++;

    if( left && ilen >= fill )
    {
        memcpy( (void *) (ctx->buffer + left),
                (void *) input, fill );
        sha1_process( ctx, ctx->buffer );
        input += fill;
        ilen  -= fill;
        left = 0;
    }

    while( ilen >= 64 )
    {
        sha1_process( ctx, input );
        input += 64;
        ilen  -= 64;
    }

    if( ilen > 0 )
    {
        memcpy( (void *) (ctx->buffer + left),
                (void *) input, ilen );
    }
}

__device__ static const unsigned char sha1_padding[64] =
{
 0x80, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
};

/*
 * SHA-1 final digest
 */
__device__ void sha1_finish( sha1_context *ctx, unsigned char output[20])
{
    unsigned long last, padn;
    unsigned long high, low;
    unsigned char msglen[8];

    high = ( ctx->total[0] >> 29 )
         | ( ctx->total[1] <<  3 );
    low  = ( ctx->total[0] <<  3 );

    PUT_ULONG_BE( high, msglen, 0 );
    PUT_ULONG_BE( low,  msglen, 4 );

    last = ctx->total[0] & 0x3F;
    padn = ( last < 56 ) ? ( 56 - last ) : ( 120 - last );

    sha1_update( ctx, (unsigned char *) sha1_padding, padn );
    sha1_update( ctx, msglen, 8 );

    PUT_ULONG_BE( ctx->state[0], output,  0 );
    PUT_ULONG_BE( ctx->state[1], output,  4 );
    PUT_ULONG_BE( ctx->state[2], output,  8 );
    PUT_ULONG_BE( ctx->state[3], output, 12 );
    PUT_ULONG_BE( ctx->state[4], output, 16 );
}

/*
 * output = SHA-1( input buffer )
 */
__device__ void sha1( unsigned char *input, int ilen, unsigned char *output )
{
    sha1_context ctx;
//    printf("hello");
    sha1_starts( &ctx );
    sha1_update( &ctx, input, ilen );
    sha1_finish( &ctx, output);
}

__global__ void sha1_kernel(unsigned char * input, unsigned char* output, unsigned int * offset, unsigned int num)
{
	const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < num)
	{
		sha1( input + offset[index], offset[index+1]-offset[index], output + index * FINGERPRINT_LEN);
	}

}

#else

#define FROM_BIG_ENDIAN(v)                                          \
 ((v & 0xff) << 24) | ((v & 0xff00) << 8) | ((v & 0xff0000) >> 8) |  \
		((v & 0xff000000) >> 24)                              \

#define LEFTROL(v, n)  (v << n) | (v >> (32 - n))

__device__ void GPU_sha1_kernel(unsigned char* data_tmp, unsigned int length_tmp,  unsigned int* md)
{

	unsigned int words[80];
	unsigned int H0 = 0x67452301,	H1 = 0xEFCDAB89, H2 = 0x98BADCFE, H3 = 0x10325476, H4 = 0xC3D2E1F0;
	unsigned int a, b, c, d, e, f, k, temp, temp2;
	unsigned int i, j;

	unsigned char add_data[MAX_CHUNK_LEN + 128];
	unsigned int kk;
	unsigned int tmp;
	unsigned long long long_tmp;
	memcpy(add_data, data_tmp, length_tmp);
	kk = length_tmp;
	if(length_tmp%64<56)
	{
		add_data[kk++]=0x80;
		int t=length_tmp%64+1;
		for(;t<56;t++)
		{
			add_data[kk++]=0x00;
		}
		tmp=length_tmp-(length_tmp%64)+64;
	}else if(length_tmp%64>56)
	{
		add_data[kk++]=0x80;
		int t=length_tmp%64+1;
		for(;t<64;t++)
		{
			add_data[kk++]=0x00;
		}
		for(t=0;t<56;t++)
		{
			add_data[kk++]=0x00;
		}
		tmp=length_tmp-(length_tmp%64)+128;
	}
	long_tmp = tmp;
	add_data[tmp-8]=(long_tmp & 0xFF00000000000000) >> 56;
	add_data[tmp-7]=(long_tmp & 0x00FF000000000000) >> 48;
	add_data[tmp-6]=(long_tmp & 0x0000FF0000000000) >> 40;
	add_data[tmp-5]=(long_tmp & 0x000000FF00000000) >> 32;
	add_data[tmp-4]=(long_tmp & 0x00000000FF000000) >> 24;
	add_data[tmp-3]=(long_tmp & 0x0000000000FF0000) >> 16;
	add_data[tmp-2]=(long_tmp & 0x000000000000FF00) >> 8;
	add_data[tmp-1]=(long_tmp & 0x00000000000000FF);

	unsigned int *data=(unsigned int*)add_data;
	unsigned int dataLen=tmp;

	for(j = 0; j < dataLen; j += 64)
	{
		a = H0;
		b = H1;
		c = H2;
		d = H3;
		e = H4;

		for (i=0; i<16; i++)
		{
			temp = *(( unsigned int*)(data + j/4+i));
			words[i] = FROM_BIG_ENDIAN(temp);

			f = (b & c) | ((~b) & d);
			k = 0x5A827999;
			temp = LEFTROL(a, 5);
			temp2 = f + e + k + words[i];
			temp = temp +temp2;
			e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp;
		}


		for (i=16; i<20; i++)
		{
			temp = (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]);
			words[i] = LEFTROL(temp, 1);
			f = (b & c) | ((~b) & d);
			temp = LEFTROL(a, 5);
			temp2 = f + e + k  + words[i];
			temp = temp + temp2;
		    e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp;
		}

		for (i=20; i<40; i++)
		{
			temp = words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16];
			words[i] = LEFTROL(temp, 1);
			f=b ^ c ^ d;
			k= 0x6ED9EBA1;
			temp = LEFTROL(a, 5);
			temp2 = f + e + k + words[i];
			temp = temp + temp2;
			e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp;
		}

		for (i=40; i<60; i++)
		{
			temp =  (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]);
		    words[i] = LEFTROL(temp, 1);
			f = (b & c) | (b & d) | (c & d);
			k = 0x8F1BBCDC;
			temp = LEFTROL(a, 5);
			temp2 = f + e + k+ words[i];
			temp = temp + temp2;
			e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp;
		}

		for (i=60; i<80; i++)
		{
			temp = (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]);
		    words[i] = LEFTROL(temp, 1);
			f = b ^ c ^ d;
			k = 0xCA62C1D6;
			temp = LEFTROL(a, 5);
			temp2 = f + e + k + words[i];
			temp = temp + temp2;
			e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp;
		}

		H0 += a;
		H1 += b;
		H2 += c;
		H3 += d;
		H4 += e;
	}

	a = H0;
	b = H1;
	c = H2;
	d = H3;
	e = H4;

	words[0] = FROM_BIG_ENDIAN(128);
	f = (b & c) | ((~b) & d);
	k = 0x5A827999;
	temp = LEFTROL(a, 5);
	temp += f + e + k + words[0];
	e = d;
	d = c;
	c = LEFTROL(b, 30);
	b = a;
	a = temp;

	for (i=1; i<15; i++)
	{
		words[i] = 0;
		f = (b & c) | ((~b) & d);
		temp = LEFTROL(a, 5);
		temp += f + e + k + words[i];
		e = d;
		d = c;
		c = LEFTROL(b, 30);
		b = a;
		a = temp;
	}


	words[15] =  dataLen*8;
	f = (b & c) | ((~b) & d);
	temp = LEFTROL(a, 5);
	temp += f + e + k + words[15];
	e = d;
	d = c;
	c = LEFTROL(b, 30);
	b = a;
	a = temp;

	for (i=16; i<20; i++)
	{
		temp = (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]);
		words[i] = LEFTROL(temp, 1);
		f = (b & c) | ((~b) & d);
		temp = LEFTROL(a, 5);
		temp += f + e + k + words[i];
		e = d;
		d = c;
		c = LEFTROL(b, 30);
		b = a;
		a = temp;
	}

	for (i=20; i<40; i++)
	{
		temp = (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]);
		words[i] = LEFTROL(temp, 1);
		f=b ^ c ^ d;
		k = 0x6ED9EBA1;
		temp = LEFTROL(a, 5);
		temp += f + e + k + words[i];
		e = d;
		d = c;
		c = LEFTROL(b, 30);
		b = a;
		a = temp;
	}

	for (i=40; i<60; i++)
	{
		temp = (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]);
		words[i] = LEFTROL(temp, 1);
		f = (b & c) | (b & d) | (c & d);
		k = 0x8F1BBCDC;
		temp = LEFTROL(a, 5);
		temp += f + e + k + words[i];
		e = d;
		d = c;
		c = LEFTROL(b, 30);
		b = a;
		a = temp;
	}

	for (i=60; i<80; i++)
	{
		temp = (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]);
		words[i] = LEFTROL(temp, 1);
		f = b ^ c ^ d;
		k = 0xCA62C1D6;
		temp = LEFTROL(a, 5);
		temp += f + e + k + words[i];
		e = d;
		d = c;
		c = LEFTROL(b, 30);
		b = a;
		a = temp;
	}

	H0 += a;
	H1 += b;
	H2 += c;
	H3 += d;
	H4 += e;


	int ct=0;
	md[ct++] =FROM_BIG_ENDIAN( H0);
	md[ct++] =FROM_BIG_ENDIAN( H1);
	md[ct++] =FROM_BIG_ENDIAN( H2);
	md[ct++] =FROM_BIG_ENDIAN( H3);
	md[ct++] =FROM_BIG_ENDIAN( H4);

}

__global__ void sha1_kernel(unsigned int *offset, unsigned char *input, unsigned char *output, unsigned int num)
{
	unsigned int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<num)
	{
		GPU_sha1_kernel(input+offset[index],offset[index+1]-offset[index],(unsigned int*)(output+index*FINGERPRINT_LEN));
	}
}
#endif

extern "C"
void GPU_sha1(unsigned char *input,unsigned char *output,unsigned int *offset,unsigned int num,unsigned int len)
{
	hipMemcpy(gpu_input_data_s,input,len*sizeof(unsigned char),hipMemcpyHostToDevice);
	hipMemcpy(gpu_offset,offset,(num+1)*sizeof(unsigned int),hipMemcpyHostToDevice);

	unsigned int threadNum=32;
	unsigned int blockNum=(unsigned int)(num+threadNum-1)/threadNum;
	dim3 grid(blockNum,1,1);
	dim3 threads(threadNum,1,1);
#ifdef MJW
	sha1_kernel<<<grid,threads>>>(gpu_input_data_s, gpu_output_data_s, gpu_offset, num);
#else
	sha1_kernel<<<grid,threads>>>(gpu_offset,gpu_input_data_s,gpu_output_data_s,num);
#endif
	hipDeviceSynchronize();

	hipMemcpy(output,gpu_output_data_s,num*FINGERPRINT_LEN,hipMemcpyDeviceToHost);
}

extern "C"
void GPU_sha1_init(unsigned int max_chunk_len,unsigned int num)
{
	hipSetDevice(0);
	hipMalloc((void**)&gpu_input_data_s, max_chunk_len*num);
	hipMalloc((void**)&gpu_output_data_s, num*FINGERPRINT_LEN);
	hipMalloc((void**)&gpu_offset, (num+1)*sizeof(unsigned int));
}

extern "C"
void GPU_sha1_destroy(void)
{
	hipFree(gpu_input_data_s);
	hipFree(gpu_output_data_s);
	hipFree(gpu_offset);
}
